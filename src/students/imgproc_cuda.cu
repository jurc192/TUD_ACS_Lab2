// Copyright 2018 Delft University of Technology
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "imgproc_cuda.hpp"

/* You could use this file to implement seperate basic effects */
Histogram getHistogramCUDA(const Image *src)
{
    std::cout << "Hello from getHistogramCUDA!\n";
    std::cout << "Image witdh: " << src->width << "\n";
    std::cout << "Image height: " << src->height << "\n";

    assert((src != nullptr));
    
    Histogram hist;

    // Host pointers
    unsigned char *h_image = &(src->raw); 
    int *h_histogram = &hist.values;

    // Device pointers
    unsigned char *d_image;
    int *d_histogram;

    // malloc 4 bytes per pixel (RGBA)
    if (hipMalloc((void **) &d_image, src->width * src->height * 4) != hipSuccess) {
        std::cout << "Failed at hipMalloc vec_a\n";
    }

    // malloc 4*256 int (4 channels, 256 values each)
    if (hipMalloc((void **) &d_histogram, 4*256*sizeof(int)) != hipSuccess) {
        std::cout << "Failed at hipMalloc vec_a\n";
    }

    // Transfer data from CPU to GPU
    hipError_t err;
    err = hipMemcpy(d_image, h_image, src->width * src->height * 4, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        std::cout << hipGetErrorString(err) << "\n";
        std::cout << "Failed at hipMemcpy at" << __FILE__ << " line: " << __LINE__ << "\n";
    }

    // kernel
 
    // Transfer data from GPU to CPU
    err = hipMemcpy(h_histogram, d_histogram, 4*256*sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        std::cout << hipGetErrorString(err) << "\n";
        std::cout << "Failed at hipMemcpy at" << __FILE__ << " line: " << __LINE__ << "\n";
    }
  
    for (int y = 0; y < src->height; y++) {
      for (int x = 0; x < src->width; x++) {
        for (int c = 0; c < 4; c++) {
          auto intensity = src->pixel(x, y).colors[c];
          hist(intensity, c)++;
        }
      }
    }

    return hist;
}